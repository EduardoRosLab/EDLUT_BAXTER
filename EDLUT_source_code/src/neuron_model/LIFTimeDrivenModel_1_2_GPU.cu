#include "hip/hip_runtime.h"
/***************************************************************************
 *                           LIFTimeDrivenModel_1_2_GPU.cu                 *
 *                           -------------------                           *
 * copyright            : (C) 2012 by Francisco Naveros                    *
 * email                : fnaveros@ugr.es                                  *
 ***************************************************************************/

/***************************************************************************
 *                                                                         *
 *   This program is free software; you can redistribute it and/or modify  *
 *   it under the terms of the GNU General Public License as published by  *
 *   the Free Software Foundation; either version 3 of the License, or     *
 *   (at your option) any later version.                                   *
 *                                                                         *
 ***************************************************************************/

#include "../../include/neuron_model/LIFTimeDrivenModel_1_2_GPU.h"
#include "../../include/neuron_model/LIFTimeDrivenModel_1_2_GPU2.h"
#include "../../include/neuron_model/VectorNeuronState.h"
#include "../../include/neuron_model/VectorNeuronState_GPU.h"

#include <iostream>
#include <cmath>
#include <string>

#include "../../include/spike/EDLUTFileException.h"
#include "../../include/spike/Neuron.h"
#include "../../include/spike/InternalSpike.h"
#include "../../include/spike/PropagatedSpike.h"
#include "../../include/spike/Interconnection.h"

#include "../../include/simulation/Utils.h"

#include "../../include/openmp/openmp.h"

#include "../../include/hipError_t.h"
//Library for CUDA
#include "hip/hip_runtime.h"
#include ""

void LIFTimeDrivenModel_1_2_GPU::LoadNeuronModel(string ConfigFile) throw (EDLUTFileException){
	FILE *fh;
	long Currentline = 0L;
	fh=fopen(ConfigFile.c_str(),"rt");
	if(fh){
		Currentline=1L;
		skip_comments(fh,Currentline);
		if(fscanf(fh,"%f",&this->eexc)==1){
			skip_comments(fh,Currentline);

			if (fscanf(fh,"%f",&this->einh)==1){
				skip_comments(fh,Currentline);

				if(fscanf(fh,"%f",&this->erest)==1){
					skip_comments(fh,Currentline);

					if(fscanf(fh,"%f",&this->vthr)==1){
						skip_comments(fh,Currentline);

						if (fscanf(fh, "%f", &this->cm) == 1 && this->cm > 0.0f){
							//Conversion from F to nF
							this->cm*=1e9;
							skip_comments(fh,Currentline);

							if (fscanf(fh, "%f", &this->texc) == 1 && this->texc > 0.0f){
								skip_comments(fh,Currentline);

								if (fscanf(fh, "%f", &this->tinh) == 1 && this->tinh > 0.0f){
									skip_comments(fh,Currentline);

									if (fscanf(fh, "%f", &this->tref) == 1 && this->tref >= 0.0f){
										skip_comments(fh,Currentline);

										if (fscanf(fh, "%f", &this->grest) == 1 && this->grest > 0.0f){
											//Conversion from S to nS
											this->grest*=1e9;
											skip_comments(fh,Currentline);

											this->State = (VectorNeuronState_GPU *) new VectorNeuronState_GPU(N_NeuronStateVariables);

										}else {
											throw EDLUTFileException(TASK_LIF_TIME_DRIVEN_MODEL_1_2_GPU_LOAD, ERROR_LIF_TIME_DRIVEN_MODEL_1_2_GREST, REPAIR_NEURON_MODEL_VALUES, Currentline, ConfigFile.c_str(), true);
										}
									}else {
										throw EDLUTFileException(TASK_LIF_TIME_DRIVEN_MODEL_1_2_GPU_LOAD, ERROR_LIF_TIME_DRIVEN_MODEL_1_2_TREF, REPAIR_NEURON_MODEL_VALUES, Currentline, ConfigFile.c_str(), true);
									}
								}else {
									throw EDLUTFileException(TASK_LIF_TIME_DRIVEN_MODEL_1_2_GPU_LOAD, ERROR_LIF_TIME_DRIVEN_MODEL_1_2_TINH, REPAIR_NEURON_MODEL_VALUES, Currentline, ConfigFile.c_str(), true);
								}
							}else {
								throw EDLUTFileException(TASK_LIF_TIME_DRIVEN_MODEL_1_2_GPU_LOAD, ERROR_LIF_TIME_DRIVEN_MODEL_1_2_TEXC, REPAIR_NEURON_MODEL_VALUES, Currentline, ConfigFile.c_str(), true);
							}
						}else {
							throw EDLUTFileException(TASK_LIF_TIME_DRIVEN_MODEL_1_2_GPU_LOAD, ERROR_LIF_TIME_DRIVEN_MODEL_1_2_CM, REPAIR_NEURON_MODEL_VALUES, Currentline, ConfigFile.c_str(), true);
						}
					}else {
						throw EDLUTFileException(TASK_LIF_TIME_DRIVEN_MODEL_1_2_GPU_LOAD, ERROR_LIF_TIME_DRIVEN_MODEL_1_2_VTHR, REPAIR_NEURON_MODEL_VALUES, Currentline, ConfigFile.c_str(), true);
					}
				}else {
					throw EDLUTFileException(TASK_LIF_TIME_DRIVEN_MODEL_1_2_GPU_LOAD, ERROR_LIF_TIME_DRIVEN_MODEL_1_2_EREST, REPAIR_NEURON_MODEL_VALUES, Currentline, ConfigFile.c_str(), true);
				}
			}else {
				throw EDLUTFileException(TASK_LIF_TIME_DRIVEN_MODEL_1_2_GPU_LOAD, ERROR_LIF_TIME_DRIVEN_MODEL_1_2_EINH, REPAIR_NEURON_MODEL_VALUES, Currentline, ConfigFile.c_str(), true);
			}
		}else {
			throw EDLUTFileException(TASK_LIF_TIME_DRIVEN_MODEL_1_2_GPU_LOAD, ERROR_LIF_TIME_DRIVEN_MODEL_1_2_EEXC, REPAIR_NEURON_MODEL_VALUES, Currentline, ConfigFile.c_str(), true);
		}

  		//INTEGRATION METHOD
		this->integrationMethod_GPU = LoadIntegrationMethod_GPU::loadIntegrationMethod_GPU((TimeDrivenNeuronModel_GPU *)this, this->GetModelID(), fh, &Currentline, N_NeuronStateVariables, N_DifferentialNeuronState, N_TimeDependentNeuronState);

		//SET TIME-DRIVEN STEP SIZE
		this->SetTimeDrivenStepSize(this->integrationMethod_GPU->elapsedTimeInSeconds);
	}else{
		throw EDLUTFileException(TASK_LIF_TIME_DRIVEN_MODEL_1_2_GPU_LOAD, ERROR_NEURON_MODEL_OPEN, REPAIR_NEURON_MODEL_NAME, Currentline, ConfigFile.c_str(), true);
	}
	fclose(fh);
}

LIFTimeDrivenModel_1_2_GPU::LIFTimeDrivenModel_1_2_GPU(string NeuronTypeID, string NeuronModelID): TimeDrivenNeuronModel_GPU(NeuronTypeID, NeuronModelID, SecondScale), eexc(0), einh(0), erest(0), vthr(0), cm(0), texc(0), tinh(0),
		tref(0), grest(0){
}

LIFTimeDrivenModel_1_2_GPU::~LIFTimeDrivenModel_1_2_GPU(void){
	DeleteClassGPU2();
}

void LIFTimeDrivenModel_1_2_GPU::LoadNeuronModel() throw (EDLUTFileException){
	this->LoadNeuronModel(this->GetModelID()+".cfg");
}

VectorNeuronState * LIFTimeDrivenModel_1_2_GPU::InitializeState(){
	return this->GetVectorNeuronState();
}


InternalSpike * LIFTimeDrivenModel_1_2_GPU::ProcessInputSpike(Interconnection * inter, double time){
	this->State_GPU->AuxStateCPU[inter->GetType()*State_GPU->GetSizeState() + inter->GetTargetNeuronModelIndex()] += inter->GetWeight();

	return 0;
}


__global__ void LIFTimeDrivenModel_1_2_GPU_UpdateState(LIFTimeDrivenModel_1_2_GPU2 ** NeuronModel_GPU2, double CurrentTime){
	(*NeuronModel_GPU2)->UpdateState(CurrentTime);
}

		
bool LIFTimeDrivenModel_1_2_GPU::UpdateState(int index, double CurrentTime){
	VectorNeuronState_GPU *state = (VectorNeuronState_GPU *) State;

	//----------------------------------------------
	if(prop.canMapHostMemory){
		LIFTimeDrivenModel_1_2_GPU_UpdateState<<<N_block,N_thread>>>(NeuronModel_GPU2, CurrentTime);
	}else{
		HANDLE_ERROR(hipMemcpy(state->AuxStateGPU,state->AuxStateCPU,this->N_TimeDependentNeuronState*state->SizeStates*sizeof(float),hipMemcpyHostToDevice));
		LIFTimeDrivenModel_1_2_GPU_UpdateState<<<N_block,N_thread>>>(NeuronModel_GPU2, CurrentTime);
		HANDLE_ERROR(hipMemcpy(state->InternalSpikeCPU,state->InternalSpikeGPU,state->SizeStates*sizeof(bool),hipMemcpyDeviceToHost));
	}
	
	if(this->GetVectorNeuronState()->Get_Is_Monitored()){
		HANDLE_ERROR(hipMemcpy(state->VectorNeuronStates,state->VectorNeuronStates_GPU,state->GetNumberOfVariables()*state->SizeStates*sizeof(float),hipMemcpyDeviceToHost));
		HANDLE_ERROR(hipMemcpy(state->LastUpdate,state->LastUpdateGPU,state->SizeStates*sizeof(double),hipMemcpyDeviceToHost));
		HANDLE_ERROR(hipMemcpy(state->LastSpikeTime,state->LastSpikeTimeGPU,state->SizeStates*sizeof(double),hipMemcpyDeviceToHost));
	} 

	HANDLE_ERROR(hipEventRecord(stop, 0)); 
	HANDLE_ERROR(hipEventSynchronize(stop));


	memset(state->AuxStateCPU,0,N_TimeDependentNeuronState*state->SizeStates*sizeof(float));

	return false;

}


enum NeuronModelOutputActivityType LIFTimeDrivenModel_1_2_GPU::GetModelOutputActivityType(){
	return OUTPUT_SPIKE;
}

enum NeuronModelInputActivityType LIFTimeDrivenModel_1_2_GPU::GetModelInputActivityType(){
	return INPUT_SPIKE;
}


ostream & LIFTimeDrivenModel_1_2_GPU::PrintInfo(ostream & out){
	out << "- Leaky Time-Driven Model 1_2: " << this->GetModelID() << endl;

	out << "\tExc. Reversal Potential: " << this->eexc << "V\tInh. Reversal Potential: " << this->einh << "V\tResting potential: " << this->erest << "V" << endl;

	out << "\tFiring threshold: " << this->vthr << "V\tMembrane capacitance: " << this->cm << "nS\tExcitatory Time Constant: " << this->texc << "s" << endl;

	out << "\tInhibitory time constant: " << this->tinh << "s\tRefractory Period: " << this->tref << "s\tResting Conductance: " << this->grest << "nS" << endl;

	return out;
}	


void LIFTimeDrivenModel_1_2_GPU::InitializeStates(int N_neurons, int OpenMPQueueIndex){

	//Select the correnpondent device. 
	HANDLE_ERROR(hipSetDevice(GPUsIndex[OpenMPQueueIndex % NumberOfGPUs]));  
	HANDLE_ERROR(hipEventCreate(&stop));
	HANDLE_ERROR(hipGetDeviceProperties( &prop, GPUsIndex[OpenMPQueueIndex % NumberOfGPUs]));


	this->State_GPU = (VectorNeuronState_GPU *) this->State;
	
	float initialization[] = {erest,0.0,0.0};
	State_GPU->InitializeStatesGPU(N_neurons, initialization, N_TimeDependentNeuronState, prop);

	//INITIALIZE CLASS IN GPU
	this->InitializeClassGPU2(N_neurons);


	InitializeVectorNeuronState_GPU2();
}




__global__ void LIFTimeDrivenModel_1_2_GPU_InitializeClassGPU2(LIFTimeDrivenModel_1_2_GPU2 ** NeuronModel_GPU2, 
		float eexc, float einh, float erest, float vthr, float cm, float texc, float tinh, float tref, float grest, 
		char const* integrationName, int N_neurons, void ** Buffer_GPU){
	if(blockIdx.x==0 && threadIdx.x==0){
		(*NeuronModel_GPU2)=new LIFTimeDrivenModel_1_2_GPU2(eexc, einh, erest, 
			vthr, cm, texc, tinh, tref, grest, integrationName, N_neurons, Buffer_GPU);
	}
}

void LIFTimeDrivenModel_1_2_GPU::InitializeClassGPU2(int N_neurons){
	hipMalloc(&NeuronModel_GPU2, sizeof(LIFTimeDrivenModel_1_2_GPU2 **));
	
	char * integrationNameGPU;
	hipMalloc((void **)&integrationNameGPU,32*4);
	HANDLE_ERROR(hipMemcpy(integrationNameGPU,integrationMethod_GPU->GetType(),32*4,hipMemcpyHostToDevice));

	this->N_thread = 128;
	this->N_block=prop.multiProcessorCount*16;
	if((N_neurons+N_thread-1)/N_thread < N_block){
		N_block = (N_neurons+N_thread-1)/N_thread;
	}
	int Total_N_thread=N_thread*N_block;

	integrationMethod_GPU->InitializeMemoryGPU(N_neurons, Total_N_thread);

	LIFTimeDrivenModel_1_2_GPU_InitializeClassGPU2<<<1,1>>>(NeuronModel_GPU2,eexc, einh, erest, vthr, 
		cm, texc, tinh, tref, grest, integrationNameGPU, N_neurons, integrationMethod_GPU->Buffer_GPU);

	hipFree(integrationNameGPU);
}



__global__ void initializeVectorNeuronState_GPU2(LIFTimeDrivenModel_1_2_GPU2 ** NeuronModel_GPU2, int NumberOfVariables, float * InitialStateGPU, float * AuxStateGPU, float * StateGPU, double * LastUpdateGPU, double * LastSpikeTimeGPU, bool * InternalSpikeGPU, int SizeStates){
	if(blockIdx.x==0 && threadIdx.x==0){
		(*NeuronModel_GPU2)->InitializeVectorNeuronState_GPU2(NumberOfVariables, InitialStateGPU, AuxStateGPU, StateGPU, LastUpdateGPU, LastSpikeTimeGPU, InternalSpikeGPU, SizeStates);
	}
}

void LIFTimeDrivenModel_1_2_GPU::InitializeVectorNeuronState_GPU2(){
	VectorNeuronState_GPU *state = (VectorNeuronState_GPU *) State;
	initializeVectorNeuronState_GPU2<<<1,1>>>(NeuronModel_GPU2, state->NumberOfVariables, state->InitialStateGPU, state->AuxStateGPU, state->VectorNeuronStates_GPU, state->LastUpdateGPU, state->LastSpikeTimeGPU, state->InternalSpikeGPU, state->SizeStates);
}


__global__ void DeleteClass_GPU2(LIFTimeDrivenModel_1_2_GPU2 ** NeuronModel_GPU2){
	if(blockIdx.x==0 && threadIdx.x==0){
		delete (*NeuronModel_GPU2); 
	}
}


void LIFTimeDrivenModel_1_2_GPU::DeleteClassGPU2(){
    DeleteClass_GPU2<<<1,1>>>(NeuronModel_GPU2);
    hipFree(NeuronModel_GPU2);
}


bool LIFTimeDrivenModel_1_2_GPU::CheckSynapseType(Interconnection * connection){
	int Type = connection->GetType();
	if (Type<N_TimeDependentNeuronState && Type >= 0){
		NeuronModel * model = connection->GetSource()->GetNeuronModel();
		//Synapse types that process input spikes 
		if (Type < N_TimeDependentNeuronState && model->GetModelOutputActivityType() == OUTPUT_SPIKE)
			return true;
		else{
			cout << "Synapses type " << Type << " of neuron model " << this->GetTypeID() << ", " << this->GetModelID() << " must receive spikes. The source model generates currents." << endl;
			return false;
		}
		//Synapse types that process input current 
	}
	else{
		cout << "Neuron model " << this->GetTypeID() << ", " << this->GetModelID() << " does not support input synapses of type " << Type << ". Just defined " << N_TimeDependentNeuronState << " synapses types." << endl;
		return false;
	}
}