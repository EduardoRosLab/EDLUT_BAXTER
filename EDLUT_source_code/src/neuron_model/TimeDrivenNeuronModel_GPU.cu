/***************************************************************************
 *                           TimeDrivenNeuronModel_GPU.cpp                 *
 *                           -------------------                           *
 * copyright            : (C) 2012 by Francisco Naveros                    *
 * email                : fnaveros@ugr.es                                  *
 ***************************************************************************/

/***************************************************************************
 *                                                                         *
 *   This program is free software; you can redistribute it and/or modify  *
 *   it under the terms of the GNU General Public License as published by  *
 *   the Free Software Foundation; either version 3 of the License, or     *
 *   (at your option) any later version.                                   *
 *                                                                         *
 ***************************************************************************/

#include "../../include/neuron_model/TimeDrivenNeuronModel_GPU.h"
#include "../../include/neuron_model/TimeDrivenNeuronModel_GPU2.h"
#include "../../include/neuron_model/TimeDrivenModel.h"
#include "../../include/neuron_model/VectorNeuronState.h"
#include "../../include/neuron_model/VectorNeuronState_GPU.h"


//Library for CUDA
#include "../../include/hipError_t.h"
#include "hip/hip_runtime.h"
#include ""

#include <string>

TimeDrivenNeuronModel_GPU::TimeDrivenNeuronModel_GPU(string NeuronTypeID, string NeuronModelID, TimeScale timeScale): TimeDrivenModel(NeuronTypeID, NeuronModelID, timeScale){
	// TODO Auto-generated constructor stub
}

TimeDrivenNeuronModel_GPU::~TimeDrivenNeuronModel_GPU() {
	delete integrationMethod_GPU;
	HANDLE_ERROR(hipEventDestroy(stop));
}

enum NeuronModelSimulationMethod TimeDrivenNeuronModel_GPU::GetModelSimulationMethod(){
	return TIME_DRIVEN_MODEL_GPU;
}

enum NeuronModelType TimeDrivenNeuronModel_GPU::GetModelType(){
	return NEURAL_LAYER;
}
