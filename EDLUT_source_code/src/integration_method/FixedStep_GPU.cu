/***************************************************************************
 *                           IntegratoinMethod_GPU.cu                      *
 *                           -------------------                           *
 * copyright            : (C) 2013 by Francisco Naveros                    *
 * email                : fnaveros@ugr.es                                  *
 ***************************************************************************/

/***************************************************************************
 *                                                                         *
 *   This program is free software; you can redistribute it and/or modify  *
 *   it under the terms of the GNU General Public License as published by  *
 *   the Free Software Foundation; either version 3 of the License, or     *
 *   (at your option) any later version.                                   *
 *                                                                         *
 ***************************************************************************/

#include "../../include/integration_method/FixedStep_GPU.h"
//#include "../../include/integration_method/FixedStep_GPU2.h"
#include "../../include/neuron_model/TimeDrivenNeuronModel_GPU.h"

#include "../../include/hipError_t.h"
//Library for CUDA
#include "hip/hip_runtime.h"
#include ""


FixedStep_GPU::FixedStep_GPU(TimeDrivenNeuronModel_GPU * NewModel, char * integrationMethodType, int N_neuronStateVariables, int N_differentialNeuronState, int N_timeDependentNeuronState):IntegrationMethod_GPU(NewModel, integrationMethodType, N_neuronStateVariables, N_differentialNeuronState, N_timeDependentNeuronState){
}

FixedStep_GPU::~FixedStep_GPU(){
}


void FixedStep_GPU::loadParameter(FILE *fh, long * Currentline, string fileName) throw (EDLUTFileException){

	skip_comments(fh, *Currentline);
	if (fscanf(fh, "%f", &elapsedTimeInSeconds) == 1){
		if (elapsedTimeInSeconds <= 0.0){
			throw EDLUTFileException(TASK_FIXED_STEP_LOAD, ERROR_FIXED_STEP_STEP_SIZE, REPAIR_FIXED_STEP, *Currentline, fileName.c_str(), true);
		}
	}
	else{
		throw EDLUTFileException(TASK_FIXED_STEP_LOAD, ERROR_FIXED_STEP_READ_STEP, REPAIR_FIXED_STEP, *Currentline, fileName.c_str(), true);
	}

	//Calculate the elapsed time size in neuron model time scale.
	elapsedTimeInNeuronModelScale = elapsedTimeInSeconds*model->GetTimeScale();
}
		


