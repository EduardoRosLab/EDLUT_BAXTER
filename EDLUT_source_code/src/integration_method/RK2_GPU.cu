#include "hip/hip_runtime.h"
/***************************************************************************
 *                           RK2_GPU.cu                                    *
 *                           -------------------                           *
 * copyright            : (C) 2013 by Francisco Naveros                    *
 * email                : fnaveros@ugr.es                                  *
 ***************************************************************************/

/***************************************************************************
 *                                                                         *
 *   This program is free software; you can redistribute it and/or modify  *
 *   it under the terms of the GNU General Public License as published by  *
 *   the Free Software Foundation; either version 3 of the License, or     *
 *   (at your option) any later version.                                   *
 *                                                                         *
 ***************************************************************************/

#include "../../include/integration_method/RK2_GPU.h"
#include "../../include/integration_method/RK2_GPU2.h"
#include "../../include/neuron_model/TimeDrivenNeuronModel_GPU2.h"

//Library for CUDA
#include "hip/hip_runtime.h"
#include ""




RK2_GPU::RK2_GPU(TimeDrivenNeuronModel_GPU * NewModel, int N_neuronStateVariables, int N_differentialNeuronState, int N_timeDependentNeuronState):FixedStep_GPU(NewModel, "RK2", N_neuronStateVariables, N_differentialNeuronState, N_timeDependentNeuronState){
}

RK2_GPU::~RK2_GPU(){
	hipFree(AuxNeuronState1);
	hipFree(AuxNeuronState2);
}

__global__ void RK2_GPU_position(void ** vector, float * integration_method_parameters_GPU, float * element1, float * element2){
	vector[0]=integration_method_parameters_GPU;
	vector[1]=element1;
	vector[2]=element2;
}
	
void RK2_GPU::InitializeMemoryGPU(int N_neurons, int Total_N_thread){
	int size=4*sizeof(float *);

	hipMalloc((void **)&Buffer_GPU, size);

	float integration_method_parameters_CPU[1];
	integration_method_parameters_CPU[0]=this->elapsedTimeInSeconds;
	float * integration_method_parameters_GPU;
	hipMalloc((void**)&integration_method_parameters_GPU, 1*sizeof(float));
	hipMemcpy(integration_method_parameters_GPU,integration_method_parameters_CPU,1*sizeof(float),hipMemcpyHostToDevice);

	hipMalloc((void**)&AuxNeuronState1, N_NeuronStateVariables*Total_N_thread*sizeof(float));
	hipMalloc((void**)&AuxNeuronState2, N_NeuronStateVariables*Total_N_thread*sizeof(float));

	RK2_GPU_position<<<1,1>>>(Buffer_GPU, integration_method_parameters_GPU, AuxNeuronState1, AuxNeuronState2);
	hipFree(integration_method_parameters_GPU);
}





